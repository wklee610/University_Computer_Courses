﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

using namespace std;          





__device__ __managed__ u32 gtime = 0;     





__device__ struct File
{
	char name[20];
	int size;               
	int last_dt;            
	int create_dt;            
};








__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,           
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,         
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE,            
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)         
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;

}







__device__ int cnt(int num){
	int count = 0;              
	while(num > 1){           
		num /= 2;                 
		count++;              
	}
	return count;       
}


//sb = super block
//b = block


__device__ void sb_change(FileSystem *fs, int s_byte, int s_idx, int b_change, int run){             
	
  if(b_change == 0 && run == 0){                              
		fs -> volume[s_byte] -= 2 << (7 - s_idx - 1);                       
		return;                             
	}

	for(int j = 0; j < b_change; j++){                    
		
    if(run == 1){                        
			fs -> volume[s_byte] += 2 << (7 - s_idx - 1);                     
		}               
		
    else          
		{                               
			fs -> volume[s_byte] -= 2 << (7 - s_idx - 1);                     
		}                 
		s_idx++;                      
		
    if(s_idx > 7){                          
			s_byte++;                   
			s_idx = 0;                                              
		}       
	}   
}           






__device__ int file_search(FileSystem *fs, char * s){                                 
	for(int i = 0; i < fs -> FCB_ENTRIES; i++){                               
    int start_addr = fs -> SUPERBLOCK_SIZE + i * fs -> FCB_SIZE;                          
		char file_name[20];                       
		int refer = 0;                                       
		
    while(true){                          
			char andy = fs -> volume[start_addr + refer];                                               
			file_name[refer] = andy;                   
			if(refer == 19){                                 
        break;              
      }                                   
			refer += 1;                                  
		}                             
		bool cd = true;                   
		
    for(int i = 0; i < sizeof(s); i++){                 
			if(file_name[i] != s[i]){                   
        cd = false;                   
      }                     
		}

		if(cd){               
      return start_addr;                        
    }                   

	}                         
	return -1;
}








__device__ int FCB_info(FileSystem *fs, int FCB_addr, int run){              
	
  u32 info = 0;               
	int refer = 0;                  
	
  
  if(run == 0){         
		refer = 20;                     
		for(int i = 0; i < 4; i++){                     
			info += (fs -> volume[FCB_addr + refer + i]) << (32 - (i + 1) * 8);                 
		}               
	}                 
	

  else if(run == 1){            
		refer = 24;                   
		for(int j = 0; j < 4; j++){                                            
			info += (fs -> volume[FCB_addr + refer + j]) << (32 - (j + 1) * 8);                                           
		}               
	}           
  
  
  else if(run == 2){                    
		refer = 28;                 
		for(int k = 0; k < 2; k++){                       
			info += (fs -> volume[FCB_addr + refer + k]) << (16 - (k + 1) * 8);                             
		}             
	}                     


  else if(run == 3){                                  
		refer = 30;                                             
		for(int l = 0; l < 2; l++){                                               
			info += (fs -> volume[FCB_addr + refer + l]) << (16 - (l + 1) * 8);                                               
		}                 
	}             


  else{           
		return -1;                
	}               


	return info;                      
}







__device__ void swap(File *xp, File *yp)                  
{       
	File reset = *xp;            
	*xp = *yp;              
	*yp =  reset;         
}       








__device__ void insertSort(File list[], int n, int type_Sort){                    
	int i, j, max_idx;                      
	if(type_Sort == LS_D){                
		printf("===sort by modified time===\n");                  
		
    for(i = 0; i < n - 1; i++){                             
			max_idx = i;            
			
      for(j = i + 1; j < n; j++){                 
				
        if(list[j].last_dt > list[max_idx].last_dt){                                
					max_idx = j;                  
        }                 
      }                     
			swap(&list[max_idx], &list[i]);                           
		}                           
		

    while(list -> name[0] != '\0'){                   
			printf("%s \n", list -> name);                    
			list++;                 
		}         
	}                 



  else if(type_Sort == LS_S){                 
			printf("===sort by size===\n");                 
			for(i = 0; i < n - 1; i++){                                               
				max_idx = i;                            
				
        for(j = i + 1; j < n; j++){               
					if(list[j].size > list[max_idx].size){                    
						max_idx = j;                  
          }                 
					
          else if(list[j].size == list[max_idx].size){                  
						if(list[j].create_dt < list[max_idx].create_dt){                    
							max_idx = j;                
            }                                                     
					}                 
				}               

				swap(&list[max_idx], &list[i]);                           
			}                           
			while(list -> name[0] != '\0'){                       
				printf("%s    %i\n", list -> name, list -> size);                      
				list++;             
		}           
	}             
}             








__device__  File * get_file(FileSystem *fs, int type_Sort){                       
	File files[1024];

  memset(files, 0, sizeof(files));                            

  int count = 0;                            
	for(int i = 0; i < fs -> FCB_ENTRIES; i++){                                   
		int start_addr = fs -> SUPERBLOCK_SIZE + i * fs -> FCB_SIZE;                      
		char file_name[20];                               
		
    for(int j = 0; j < 20; j++){                      
      file_name[j] = fs -> volume[start_addr + j];                          
    }               
		
    if(file_name[0] != '\0'){                 
			File file;                                      
			file.last_dt = FCB_info(fs, start_addr, 2);                   
			file.create_dt = FCB_info(fs, start_addr, 3);                 
			
      for(int j = 0; j < 20; j++){                              
        file.name[j] = file_name[j];                              
      }

			file.size = FCB_info(fs, start_addr, 1);                
			files[count] = file;                       
			count++;            
		}             
	}                 


  insertSort(files, count, type_Sort);                                       


  return files;                                     
}               



__device__ u32 change_addr(FileSystem *fs, int s_byte, int s_idx){             
	if(s_byte > 0){               
    return (s_byte - 1) * 8 * 32 + s_idx * 32 + fs -> FILE_BASE_ADDRESS;        
  }           

  return s_idx * 32 + fs -> FILE_BASE_ADDRESS;          
}                         





__device__ void FCB_update(FileSystem *fs, u32 addr, int size, u32 fp, int run){  
	int start_addr = fp + 20;                           
	int reset = 0;                        
	
  for(int i = 0; i < 4; i++){     
		reset = (addr - (u32)(reset << (32 - 8 * i))) >> (32 - 8 * (i + 1));              
		fs -> volume[start_addr + i] = reset;           
	}                                                   
	
  
  reset = 0;                        
	start_addr = fp + 24;             
	
  
  for(int j = 0; j < 4; j++){                   
		reset = (size - (u32)(reset << (32 - 8 * j))) >> (32 - 8 * (j + 1));            
		fs -> volume[start_addr + j] = reset;   
	}               
	
  
  gtime++;              
	reset = 0;            
	start_addr = fp + 28;               
	
  
  for(int k = 0; k < 2; k++){                   
		reset = (gtime - (u32)(reset << (16 - 8 * k))) >> (16 - 8 * (k + 1));                     
		fs -> volume[start_addr + k] = reset;               
	}                                                                       
	
  
  if(run == 1){                       
		reset = 0;                
		start_addr = fp + 30;             
		
    for(int l = 0; l < 2; l++){                      
			reset = (gtime - (u32)(reset << (16 - 8 * l))) >> (16 - 8 * (l + 1));                       
			fs -> volume[start_addr + l] = reset;                 
		}           
	}             
}           
























__device__ u32 fs_open(FileSystem *fs, char *s, int op){                  
	/* Implement open operation here */
	
  

  u32 addr = NULL;                  
	int start_addr = file_search(fs, s);                              
	
  
  if(start_addr != -1){                                               
    return start_addr;                        
  }                                   

  if(op  ==   G_READ){                                                     
		printf("No file");                                             
		return -1;                                                              
	}                 

	bool cd = false;                                 
	int b_idx = 0;                                                  
	
  for(int m = 0; m < fs -> SUPERBLOCK_SIZE; m++){                         
		if(cd){                         
      break;                        
    }               

		int total = fs -> volume[m];                                   
		
    if(total != 255){                                         
			for(int i = 128;  i > 1;  i /= 2){              
				if(total / i == 0){              
					cd = true;       
					fs -> volume[m] += i;           
					addr = change_addr(fs, m, b_idx);             
					break;                
				}

        else{                   
					total -= i;                 
					b_idx++;                
				}             
			}               
		}             
	}               





	for(int i = 0; i < fs -> FCB_ENTRIES; i++){                   
		int start_addr = fs -> SUPERBLOCK_SIZE + i * fs -> FCB_SIZE;                  
		
    if(fs -> volume[start_addr] != '\0'){           
      continue;                 
    }                         
    
		int index = 0;                             

    while(s[index] != '\0'){                            
	  		fs -> volume[start_addr + index] = s[index];                    
			  index++;                    
		}                                    

		FCB_update(fs, addr, 0, start_addr, 1);                     
		return start_addr;                  
	}                             

	return -1;                  

}





__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp){
	/* Implement read operation here */
	int addr = FCB_info(fs, fp, 0);                       
	if(addr == -1){             
    return;             
  }                     
	for(int i = 0; i < size; i++){                    
		output[i] = fs -> volume[addr + i];           
	}         

  gtime++;                  
}                               










__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp){                
	/* Implement write operation here */
	int b_use = size  % 32 ==  0 ? size / 32 :  size / 32 +  1;             
	int addr = FCB_info(fs, fp, 0);                     
	
  if(addr == -1){       
    return;         
  }                         
	

  int b_addr = (addr - fs -> FILE_BASE_ADDRESS) / 32;                            
	int s_byte = b_addr / 8;                          
	int s_idx = b_addr % 8;                   
	int first_size = FCB_info(fs, fp, 1);                         
                            
  first_size = first_size % 32 == 0 ? first_size / 32 : first_size / 32 + 1;                
	sb_change(fs, s_byte, s_idx, first_size, 0);            


	int b_cnt = 0;                
	bool first_cd =  true;         
	int index = 0;                  
	
  
  
  
  while(b_cnt < b_use){                                             
		int total = fs -> volume[index];                    
		for(int i = 128; i > 1; i /= 2){                  
			if(total / i == 0){                       
				if(first_cd){                     
					s_idx = 7 - cnt(i);                   
					s_byte = index;                   
					first_cd = false;                 
				}                     

				b_cnt++;                          
				if(b_cnt == b_use){                               
          break;              
        }           
			}               
			

      else{             
				first_cd = true;              
				b_cnt = 0;              
				total -= i;           
			}               

		}                 
		index++;                      
		
    if(index > fs -> SUPERBLOCK_SIZE - 1 && b_cnt < b_use){               
      return -1;            
    }                     
	}                   
	
  sb_change(fs, s_byte, s_idx, b_use, 1);                                  
	addr = change_addr(fs, s_byte, s_idx);                         
	
  FCB_update(fs, addr, size, fp, 0);                              
	for(int k = 0; k < size; k++){            
    fs -> volume[addr + k] = input[k];              
  }                       
}                   





__device__ void fs_gsys(FileSystem *fs, int op){                
	/* Implement LS_D and LS_S operation here */  
  get_file(fs, op);               
}             








__device__ void fs_gsys(FileSystem *fs, int op, char *s){                         
	/* Implement rm operation here */
	if(op != RM){                                                     
    return;                         
  }                     
	
  int FCB_file_addr = file_search(fs, s);                             
	
  if(FCB_file_addr == -1){                
		printf("No file\n");                      
		return;                 
	}             

	int space_addr = FCB_info(fs, FCB_file_addr, 0);                            
	int size = FCB_info(fs, FCB_file_addr, 1);                   
	
  
  for(int i = 0; i < size; i++){                                    
    fs -> volume[space_addr + i] = 0;             
  }                       
	
  size =  size  %  32 == 0  ?  size / 32 : size  /  32  + 1;                                              
	
  int b_addr = (space_addr - fs -> FILE_BASE_ADDRESS) / 32;                     
	int s_byte = b_addr / 8;              
	int s_idx = b_addr % 8;                             
	sb_change(fs, s_byte, s_idx, size, 0);                  

	for(int j = 0; j < fs -> FCB_SIZE; j++){                
    fs -> volume[FCB_file_addr + j] = 0;                
  }           
}                 
