﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>






__device__ void init_invert_page_table(VirtualMemory *vm) {

  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    vm->invert_page_table[i] = 0x80000000; // invalid := MSB is 1
    vm->invert_page_table[i + vm->PAGE_ENTRIES] = i;
  }
}

__device__ void vm_init(VirtualMemory *vm, uchar *buffer, uchar *storage,
                        u32 *invert_page_table, int *pagefault_num_ptr,
                        int PAGESIZE, int INVERT_PAGE_TABLE_SIZE,
                        int PHYSICAL_MEM_SIZE, int STORAGE_SIZE,
                        int PAGE_ENTRIES) {
  // init variables
  vm->buffer = buffer;
  vm->storage = storage;
  vm->invert_page_table = invert_page_table;
  vm->pagefault_num_ptr = pagefault_num_ptr;

  // init constants
  vm->PAGESIZE = PAGESIZE;
  vm->INVERT_PAGE_TABLE_SIZE = INVERT_PAGE_TABLE_SIZE;
  vm->PHYSICAL_MEM_SIZE = PHYSICAL_MEM_SIZE;
  vm->STORAGE_SIZE = STORAGE_SIZE;
  vm->PAGE_ENTRIES = PAGE_ENTRIES;

  // before first vm_write or vm_read
  init_invert_page_table(vm);
}




__device__ int idx_search(VirtualMemory *vm, int idx_page){                 
	int LRU_SET = vm -> invert_page_table[vm -> PAGE_ENTRIES];                    
	int idx = 0;                    
	
  for(int i = 0; i < vm -> PAGE_ENTRIES; i++){      
		if(idx_page == vm -> invert_page_table[i]){     
			idx = i;          
			vm -> invert_page_table[idx + vm -> PAGE_ENTRIES] ++;         
			return idx;         
		}
	}
	
  (*vm -> pagefault_num_ptr)++;                  
	
  for(int j = 0; j < vm->PAGE_ENTRIES; j++){      
		if(vm -> invert_page_table[j + vm -> PAGE_ENTRIES] < LRU_SET){    
			idx = j;              
			LRU_SET = vm -> invert_page_table[j + vm -> PAGE_ENTRIES];       
		}
	}         
	
  for(int l = 0; l < vm->PAGESIZE; l++){      
		vm -> storage[vm -> invert_page_table[idx] * vm -> PAGESIZE] = vm -> buffer[idx * vm -> PAGESIZE + l];        
		vm -> buffer[idx * vm -> PAGESIZE + l] = vm -> storage[idx_page * vm -> PAGESIZE + l];        
	}     
	
  vm -> invert_page_table[idx + vm -> PAGE_ENTRIES] ++;       
	vm -> invert_page_table[idx] = idx_page;        
	return idx;                 
}     




__device__ uchar vm_read(VirtualMemory *vm, u32 addr){        
	u32 offset = addr % vm -> PAGESIZE;             
  
  int idx_page = addr / vm -> PAGESIZE;               
	int idx = idx_search(vm, idx_page);             
	
  return vm -> buffer[(u32)(idx * vm -> PAGESIZE) + offset];                         
}




__device__ void vm_write(VirtualMemory *vm, u32 addr, uchar value){         
	u32 offset = addr % vm -> PAGESIZE;                   

  int idx_page = addr / vm -> PAGESIZE;                         
	int idx = idx_search(vm, idx_page);                         
	
  vm -> buffer[(u32)(idx * vm -> PAGESIZE) + offset] = value;               
	vm -> storage[addr] = vm -> buffer[(u32)(idx * vm -> PAGESIZE) + offset];        

}




__device__ void vm_snapshot(VirtualMemory *vm, uchar *results, int offset,        
                            int input_size) {
	for(int i = offset; i < offset + input_size; i++){  
		results[i] = vm_read(vm, i);      

    } 
  } 
